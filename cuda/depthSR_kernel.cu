#include "hip/hip_runtime.h"
#include "depthSR_kernel.cuh"

__global__ void memcpy_float(int width, float* dst, float* src)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    dst[y * width + x] = src[y * width + x];
}

__global__ void extractCorrelation_kernel(ContextDepthSR *context, uint8_t *LR_GrayBorder, float *LR_DepthBorder, float *GD_correlation)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pixel = y + x * blockDim.y * gridDim.y;

    int width = context->width;
    int height = context->height;
    int scale_w = context->scale_w;
    int scale_h = context->scale_h;
    int s_width = width / scale_w;
    int s_height = height / scale_h;


    int win_size = 3; //3x3 windows
    float sumPixG = 0.0, sumPixG2 = 0.0;
    float sumPixD = 0.0, sumPixD2 = 0.0;
    float sumPixGD = 0.0;
    for (int m = -win_size / 2; m <= win_size / 2; m++){
        int i = x + 1 + m;
        i = (i > 0 ? (i < s_height + 2 ? i : s_height + 1) : 0);  //make sure the index (x+n, y+m) is located in the image.
        for (int n = -win_size / 2; n <= win_size / 2; n++){
            int j = y + 1 + n;
            j = (j > 0 ? (j < s_width + 2 ? j : s_width + 1) : 0);
            uint8_t a1 = LR_GrayBorder[i * (s_width + 2) + j];

            float a2 = LR_DepthBorder[i * (s_width + 2) + j];
            sumPixG += a1;
            sumPixD += a2;
            sumPixG2 += (a1 * a1);
            sumPixD2 += (a2 * a2);
            sumPixGD += (a1 * a2);
        }//end for n
    }//end for m
    float meanPixG = sumPixG / (win_size * win_size);//EX
    float meanPixD = sumPixD / (win_size * win_size);
    float meanPixG2 = sumPixG2 / (win_size * win_size); //E(X^2)
    float meanPixD2 = sumPixD2 / (win_size * win_size);
    float meanPixGD = sumPixGD / (win_size * win_size);
    float CA = meanPixGD - meanPixG * meanPixD;
    float variancePixG = meanPixG2 - meanPixG * meanPixG;
    float variancePixD = meanPixD2 - meanPixD * meanPixD;
    CA /= sqrt(variancePixG * variancePixD);
    GD_correlation[pixel] = CA;
}


ContextDepthSR* create_context(ContextDepthSR* context)
{
    ContextDepthSR *cu_context;
    hipMallocManaged((void**)&cu_context, sizeof(ContextDepthSR));

    hipMemcpy(&cu_context->width, &context->width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&cu_context->height, &context->height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&cu_context->scale_w, &context->scale_w, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&cu_context->scale_h, &context->scale_h, sizeof(int), hipMemcpyHostToDevice);

    return cu_context;
}
void extractCorrelation(ContextDepthSR *context, uint8_t *LR_Gray, float *LR_Depth, float *GD_correlation)
{
    int width = context->width;
    int height = context->height;
    int scale_w = context->scale_w;
    int scale_h = context->scale_h;
    int s_width = width / scale_w;
    int s_height = height / scale_h;

 //get the padding border image
    int border = 2;
    size_t size = (s_width + border) * (s_height + border);
    uint8_t *LR_GrayBorder = (uint8_t*)calloc(size, sizeof(uint8_t));
    float *LR_DepthBorder = (float*)calloc(size, sizeof(float));
    for(int i = 1; i < s_height + 1; i++){
        for(int j = 1; j < s_width + 1; j++){
            int pixel_b = i * (s_width + border) + j;
            int pixel = (i-1) * s_width + (j-1);
            LR_GrayBorder[pixel_b] = LR_Gray[pixel];
            LR_DepthBorder[pixel_b] = LR_Depth[pixel];
        }
    }//CENTER
    for(int i = 1; i < s_width + 1; i++){
        LR_GrayBorder[i] = LR_Gray[i - 1];
        LR_DepthBorder[i] = LR_Depth[i - 1]; //TOP

        int pixel_b = (s_width + border) * (s_height + 1) + i;
        int pixel = s_width * (s_height - 1) + i - 1;
        LR_GrayBorder[pixel_b] = LR_Gray[pixel];
        LR_DepthBorder[pixel_b] = LR_Depth[pixel];//BOTTOM
    }
    for(int j = 1; j < s_height + 1; j++){
        LR_GrayBorder[j * (s_width + border)] = LR_GrayBorder[j * (s_width + border) + 1];
        LR_DepthBorder[j * (s_width + border)] = LR_DepthBorder[j * (s_width + border) + 1];//LEFT

        LR_GrayBorder[(j + 1) * (s_width + border) - 1] = LR_GrayBorder[(j + 1) * (s_width + border) - 2];
        LR_DepthBorder[(j + 1) * (s_width + border) - 1] = LR_DepthBorder[(j + 1) * (s_width + border) - 2];//RIGHT
    }

    //the four corner value
    int pixel_corner;
    LR_GrayBorder[0] = (LR_GrayBorder[1] + LR_GrayBorder[(s_width + border)]) / 2;
    LR_DepthBorder[0] = (LR_DepthBorder[1] + LR_DepthBorder[(s_width + border)]) / 2;

    pixel_corner = (s_width + border) - 1;
    LR_GrayBorder[pixel_corner] = (LR_GrayBorder[(pixel_corner - 1)] + LR_GrayBorder[(pixel_corner + s_width + border)]) / 2;
    LR_DepthBorder[pixel_corner] = (LR_DepthBorder[(pixel_corner - 1)] + LR_DepthBorder[(pixel_corner + s_width + border)]) / 2;

    pixel_corner = ((s_height + border) - 1) * (s_width + border);
    LR_GrayBorder[pixel_corner] = (LR_GrayBorder[(pixel_corner + 1)] + LR_GrayBorder[(pixel_corner - s_width - border)]) / 2;
    LR_DepthBorder[pixel_corner] = (LR_DepthBorder[(pixel_corner + 1)] + LR_DepthBorder[(pixel_corner - s_width - border)]) / 2;

    pixel_corner = (s_height + border) * (s_width + border) - 1;
    LR_GrayBorder[pixel_corner] = (LR_GrayBorder[(pixel_corner - 1)] + LR_GrayBorder[(pixel_corner - s_width - border)]) / 2;
    LR_DepthBorder[pixel_corner] = (LR_DepthBorder[(pixel_corner - 1)] + LR_DepthBorder[(pixel_corner - s_width - border)]) / 2;

    dim3 blocks(s_height / THREADS_PER_BLOCK, s_width / ThreadsPerBlock);
    dim3 threads(THREADS_PER_BLOCK, ThreadsPerBlock);


    uint8_t* cu_LR_GrayBorder;
    float* cu_LR_DepthBorder;
    float* cu_GD_correlation;

    hipMalloc((void**)&cu_LR_GrayBorder, (s_height + border) * (s_width + border) * sizeof(uint8_t));
    hipMalloc((void**)&cu_LR_DepthBorder, (s_height + border) * (s_width + border) * sizeof(float));
    hipMemcpy(cu_LR_GrayBorder, LR_GrayBorder, (s_height + border) * (s_width + border) * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(cu_LR_DepthBorder, LR_DepthBorder, (s_height + border) * (s_width + border) * sizeof(float), hipMemcpyHostToDevice);

    hipMallocManaged((void**)&cu_GD_correlation, s_width * s_height * sizeof(float));

    extractCorrelation_kernel<<<blocks, threads>>>(context, cu_LR_GrayBorder, cu_LR_DepthBorder, cu_GD_correlation);

    //memcpy_float<<<blocks, threads>>>(s_width, gray_variance, cu_gray_variance);
    //memcpy_float<<<blocks, threads>>>(s_width, depth_variance, cu_depth_variance);
    hipMemcpy(GD_correlation, cu_GD_correlation, s_width * s_height * sizeof(float), hipMemcpyDeviceToHost);
    //memcpy(GD_correlation, cu_GD_correlation, s_width * s_height * sizeof(float));

    hipFree(cu_LR_GrayBorder);
    hipFree(cu_LR_DepthBorder);
    hipFree(cu_GD_correlation);
}